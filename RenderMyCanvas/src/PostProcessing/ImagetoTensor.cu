#include "hip/hip_runtime.h"
#include"ImagetoTensor.cuh"
#include <glm/glm.hpp>
#include<iostream>
 
// CUDA �˺���������ִ������ڲ�ֵ
__global__ void NearestNeighborKernel(const uint32_t* d_img_data, int old_width, int old_height, uint32_t* d_scaled_img_data, int new_width, int new_height)
{
    // ��ȡ�̵߳�ȫ������
    int new_x = blockIdx.x * blockDim.x + threadIdx.x;
    int new_y = blockIdx.y * blockDim.y + threadIdx.y;

    // ȷ���߳���������ͼ��ķ�Χ��
    if (new_x < new_width && new_y < new_height)
    {
        // ����ԭͼ���е�����
        int orig_x = new_x / 2;
        int orig_y = new_y / 2;
        int orig_index = orig_y * old_width + orig_x;

        // ������ͼ���е�����
        int new_index = new_y * new_width + new_x;

        // ��ԭͼ�������ֵ���Ƶ���ͼ����
        d_scaled_img_data[new_index] = d_img_data[orig_index];
    }
}


// ���� CUDA �˺�����������
void NearestNeighborScaleCUDA(uint32_t* img_data, int old_width, int old_height, uint32_t* scaled_img_data)
{
    uint32_t* d_img_data, *d_scaled_img_data;

    // ���豸�Ϸ����ڴ�
    hipMalloc(&d_img_data, old_width * old_height * sizeof(uint32_t));
    hipMalloc(&d_scaled_img_data, old_width * 2 * old_height * 2 * sizeof(uint32_t));

    // �����ݴ��������Ƶ��豸
    hipMemcpy(d_img_data, img_data, old_width * old_height * sizeof(uint32_t), hipMemcpyHostToDevice);


    int new_width = old_width * 2;
    int new_height = old_height * 2;

    // ���� CUDA �߳̿������Ĵ�С
    dim3 blockSize(16, 16); // ÿ���߳̿� 16x16 ���߳�
    dim3 gridSize((new_width + blockSize.x - 1) / blockSize.x, (new_height + blockSize.y - 1) / blockSize.y);

    // ���� CUDA �˺���
    NearestNeighborKernel << <gridSize, blockSize >> > (d_img_data, old_width, old_height, d_scaled_img_data, new_width, new_height);

    // ��������豸���ƻ�����
    hipMemcpy(scaled_img_data, d_scaled_img_data, new_width * new_height * sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    // �ͷ��豸�ϵ��ڴ�
    hipFree(d_img_data);
    hipFree(d_scaled_img_data);
}


void ImageToTensor(const uint32_t* img_data, float* tensor_data, int width, int height) 
{
    uint32_t* d_img_data;
    hipMalloc(&d_img_data, width * height * sizeof(uint32_t));
    //std::cout << "start convert" << std::endl;
    hipMemcpy(d_img_data, img_data, width * height * sizeof(uint32_t), hipMemcpyHostToDevice);


    // �趨CUDA������߳̿��С
    dim3 block_size(16, 16);
    dim3 grid_size((width*2 + block_size.x - 1) / block_size.x, (height*2 + block_size.y - 1) / block_size.y);

    // ���� CUDA �˺���
    ImageToTensorKernel << <grid_size, block_size >> > (d_img_data, tensor_data, width, height);

    // �ȴ�CUDA���
    hipDeviceSynchronize();

    // �ͷŷ�����ڴ�
    hipFree(d_img_data);

}

// CUDA �˺�������������ͼ������
__global__ void ImageToTensorKernel(const uint32_t* img_data, float* tensor_data, int width, int height) {
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.x * blockDim.x + threadIdx.x;

    if (h < height && w < width) {
        int index = h * width + w;
        uint32_t pixel = img_data[index];

        // RGBAת��Ϊ������ɫֵ
        float r = ((pixel >> 24) & 0xFF) / 255.0f;
        float g = ((pixel >> 16) & 0xFF) / 255.0f;
        float b = ((pixel >> 8) & 0xFF) / 255.0f;

        int base_index = (h * width + w) * 3;
        tensor_data[base_index] = r;
        tensor_data[base_index + 1] = g;
        tensor_data[base_index + 2] = b;
    }
}

void convertTensorToImage(const float* data_ptr, uint32_t* img_data, int width, int height)
{
    // ���豸�Ϸ����ڴ�
       //float* d_tensor_data;
       uint32_t* d_img_data;
       //hipMalloc(&d_tensor_data, sizeof(float)* (width* height * 3));
       hipMalloc(&d_img_data, sizeof(uint32_t)* (width* height));

       // �����ݴ��������Ƶ��豸
       //hipMemcpy(d_tensor_data, data_ptr, sizeof(float)* (width* height * 3), hipMemcpyHostToDevice);

       // 4. ���� CUDA ����Ϳ��ά��
       dim3 blockSize(16, 16); // ÿ�����е��߳���
       dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y); // ����ߴ�

       // 5. ���� CUDA �ں�
       convertTensorToImageKernel << <gridSize, blockSize >> > (data_ptr, d_img_data, height, width);
       hipDeviceSynchronize(); // ȷ���ں�ִ�����

       // 6. ���豸�������ݻ�����
       hipMemcpy(img_data, d_img_data, sizeof(uint32_t)* (width* height), hipMemcpyDeviceToHost);

       // 8. �ͷ��豸�ڴ�
       //hipFree(d_tensor_data);
       hipFree(d_img_data);

}

__global__ void convertTensorToImageKernel(const float* tensor_data, uint32_t* img_data, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����ÿ���̵߳� x ����
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����ÿ���̵߳� y ����

    if (x < width && y < height) {
        // ���㵱ǰ���ص�����
        int index = y * width + x;

        // ��ȡ��ɫͨ��
        float r = tensor_data[index]; // Red
        float g = tensor_data[1 * height * width + index]; // Green
        float b = tensor_data[2 * height * width + index]; // Blue
        float a = 1.0f; // Alpha ֵ

        // ����ɫת��Ϊ uint32_t
        uint32_t uint_r = static_cast<uint32_t>(r * 255.0f);
        uint32_t uint_g = static_cast<uint32_t>(g * 255.0f);
        uint32_t uint_b = static_cast<uint32_t>(b * 255.0f);
        uint32_t uint_a = static_cast<uint32_t>(a * 255.0f);

        img_data[index] = (uint_a << 24) | (uint_b << 16) | (uint_g << 8) | uint_r; // ֱ�����ں������� RGBA ֵ
    }
}
