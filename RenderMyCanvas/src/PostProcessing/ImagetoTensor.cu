#include "hip/hip_runtime.h"
#include"ImagetoTensor.cuh"
#include <glm/glm.hpp>
#include<iostream>
 
void ImageToTensor(const uint32_t* img_data, float* tensor_data, int width, int height) 
{
    uint32_t* d_img_data;
    hipMalloc(&d_img_data, width * height * sizeof(uint32_t));
    //std::cout << "start convert" << std::endl;
    hipMemcpy(d_img_data, img_data, width * height * sizeof(uint32_t), hipMemcpyHostToDevice);

    float *d_tensor_data;
    hipMalloc(&d_tensor_data, width * height * 3 * sizeof(float));

    // �趨CUDA������߳̿��С
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    // ���� CUDA �˺���
    ImageToTensorKernel << <grid_size, block_size >> > (d_img_data, d_tensor_data, width, height);

    // �ȴ�CUDA���
    hipDeviceSynchronize();

    // ��������豸���ƻ�����
    hipMemcpy(tensor_data, d_tensor_data, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost);

    // �ͷŷ�����ڴ�
    hipFree(d_img_data);
    hipFree(d_tensor_data);
    //std::cout << "end convert" << std::endl;    

}

// CUDA �˺�������������ͼ������
__global__ void ImageToTensorKernel(const uint32_t* img_data, float* tensor_data, int width, int height) {
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.x * blockDim.x + threadIdx.x;

    if (h < height && w < width) {
        int index = h * width + w;
        uint32_t pixel = img_data[index];

        // RGBAת��Ϊ������ɫֵ
        float r = ((pixel >> 24) & 0xFF) / 255.0f;
        float g = ((pixel >> 16) & 0xFF) / 255.0f;
        float b = ((pixel >> 8) & 0xFF) / 255.0f;

        int base_index = (h * width + w) * 3;
        tensor_data[base_index] = r;
        tensor_data[base_index + 1] = g;
        tensor_data[base_index + 2] = b;
    }
}

void convertTensorToImage(const float* data_ptr, uint32_t* img_data, int width, int height)
{
    // ���豸�Ϸ����ڴ�
       float* d_tensor_data;
       uint32_t* d_img_data;
       hipMalloc(&d_tensor_data, sizeof(float)* (width* height * 3));
       hipMalloc(&d_img_data, sizeof(uint32_t)* (width* height));

       // �����ݴ��������Ƶ��豸
       hipMemcpy(d_tensor_data, data_ptr, sizeof(float)* (width* height * 3), hipMemcpyHostToDevice);

       // 4. ���� CUDA ����Ϳ��ά��
       dim3 blockSize(16, 16); // ÿ�����е��߳���
       dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y); // ����ߴ�

       // 5. ���� CUDA �ں�
       convertTensorToImageKernel << <gridSize, blockSize >> > (d_tensor_data, d_img_data, height, width);
       hipDeviceSynchronize(); // ȷ���ں�ִ�����

       // 6. ���豸�������ݻ�����
       hipMemcpy(img_data, d_img_data, sizeof(uint32_t)* (width* height), hipMemcpyDeviceToHost);

       // 8. �ͷ��豸�ڴ�
       hipFree(d_tensor_data);
       hipFree(d_img_data);

}

__global__ void convertTensorToImageKernel(const float* tensor_data, uint32_t* img_data, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // ����ÿ���̵߳� x ����
    int y = blockIdx.y * blockDim.y + threadIdx.y; // ����ÿ���̵߳� y ����

    if (x < width && y < height) {
        // ���㵱ǰ���ص�����
        int index = y * width + x;

        // ��ȡ��ɫͨ��
        float r = tensor_data[index]; // Red
        float g = tensor_data[1 * height * width + index]; // Green
        float b = tensor_data[2 * height * width + index]; // Blue
        float a = 1.0f; // Alpha ֵ

        // ����ɫת��Ϊ uint32_t
        uint32_t uint_r = static_cast<uint32_t>(r * 255.0f);
        uint32_t uint_g = static_cast<uint32_t>(g * 255.0f);
        uint32_t uint_b = static_cast<uint32_t>(b * 255.0f);
        uint32_t uint_a = static_cast<uint32_t>(a * 255.0f);

        img_data[index] = (uint_a << 24) | (uint_b << 16) | (uint_g << 8) | uint_r; // ֱ�����ں������� RGBA ֵ
    }
}
