#include "hip/hip_runtime.h"
#include"ImagetoTensor.cuh"
#include<hip/hip_runtime.h>

// CUDA �˺�������������ͼ������
__global__ void ConvertImageToTensorKernel(const uint32_t* img_data, float* output_tensor, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // ������ԭʼ�����е�ƫ����
    int index = y * width + x;
    uint32_t pixel = img_data[index];

    // �����豸�˵� ConvertFromRGBA ����
    glm::vec4 color = Utils::ConvertFromRGBA(pixel);

    // �洢�����������
    int tensor_idx = (y * width + x); // �� GPU �ϴ�������Ϊ�������ڴ�ռ�
    output_tensor[tensor_idx] = color.r;           // Red
    output_tensor[tensor_idx + width * height] = color.g; // Green
    output_tensor[tensor_idx + 2 * width * height] = color.b; // Blue
}